#include "hip/hip_runtime.h"
#include <stdio.h>
#include <malloc.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>

#define N 1024 // 正方行列のサイズを指定（N×N）
#define BLOCK 16 // ブロックのサイズを指定

__global__ void
matrixMul(int* inMatA, int* inMatB, int* inMatC);

int main(int argc, char** argv){

    // 行列のサイズをバイト単位で算出
    int matrixSize = sizeof(unsigned int) * N * N;

    // ホスト側の行列変数設定
    int* hMatA;
    int* hMatB;
    int* hMatC;

    // 行列変数のメモリ確保
    hMatA = (int*)malloc(matrixSize);
    hMatB = (int*)malloc(matrixSize);

    // 初期値設定
    int col, row;
    for (col = 0; col < N; col++){
        for (row = 0; row < N; row++){
            hMatA[col * N + row] = rand() % (N * N);
            hMatB[col * N + row] = rand() % (N * N);
        }
    }

    // デバイス側の行列変数設定
    int* dMatA;
    int* dMatB;
    int* dMatC;

    // デバイスメモリ領域の確保
    hipMalloc((void**)&dMatA, matrixSize);
    hipMalloc((void**)&dMatB, matrixSize);
    hipMalloc((void**)&dMatC, matrixSize);

    ////////////////////////////////////////////////////////////////////////
    StopWatchInterface *timer=NULL;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);
    sdkStartTimer(&timer);

    // ホストからデバイスへの変数の受け渡し
    hipMemcpy(dMatA, hMatA, matrixSize, hipMemcpyHostToDevice);
    hipMemcpy(dMatB, hMatB, matrixSize, hipMemcpyHostToDevice);

    // ブロックサイズとグリッドサイズの設定
    dim3 block(BLOCK, BLOCK);
    dim3 grid( N / BLOCK, N / BLOCK);

    // カーネルの起動
    matrixMul<<<grid, block>>>(dMatA, dMatB, dMatC);
    hipDeviceSynchronize();

    // 結果の領域確保とデバイス側からのメモリ転送
    hMatC = (int*)malloc(matrixSize);
    hipMemcpy(hMatC, dMatC, matrixSize, hipMemcpyDeviceToHost);

    ////////////////////////////////////////////////////////////////////////
    sdkStopTimer(&timer);
    float time = sdkGetTimerValue(&timer);
    sdkDeleteTimer(&timer);
    printf("Processing time: %f (msec)\n", time);

    // ホスト・デバイスメモリの解放
    free(hMatA);
    free(hMatB);
    free(hMatC);
    hipFree(dMatA);
    hipFree(dMatB);
    hipFree(dMatC);

    // 終了処理
    hipDeviceReset();
    //cutilExit(argc, argv);
}

__global__ void matrixMul(int* inMatA, int* inMatB, int* inMatC)
  {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int scan;
  int target = 0;

  // 行列の演算を行う
  for (scan = 0; scan < N; scan++) {
      target += inMatA[col * N + scan] * inMatB[scan * N + row];
      __syncthreads();
  }

  inMatC[col * N + row] = target;
}
