#include "hip/hip_runtime.h"
#include<stdio.h>
#include<malloc.h>
#include<stdlib.h>
#include<hip/hip_runtime_api.h>

#define N 1024
#define BLOCK 16

__global__ void matrixMul(int* inMatA, int* inMatB, int* inMatC);

int main(int argc,char** argv){

  int matrixSize=sizeof((unsigned int)*N*N);

  int hMatA;
  int hMatB;
  int hMatC;

  hMatA=(int*)malloc(matrixSize);
  hMatB=(int*)malloc(matrixSize);

  int col,row;
  for(col=0;col<N;col++){
    for(row=0;row<N;row++){
      hMatA[col*N+row]=rand()%(N*N);
      hMatB[col*N+row]=rand()%(N*N);
    }
  }

  int dMatA;
  int dMatB;
  int dMatC;

  hipMalloc((void**)&dMatA,matrixSize);
  hipMalloc((void**)&dMatB,matrixSize);
  hipMalloc((void**)&dMatC,matrixSize);

  hipMemcpy(dMatA,hMatA,matrixSize,hipMemcpyHostToDevice);
  hipMemcpy(dMatB,hMatB,matrixSize,hipMemcpyHostToDevice);

  dim3 block(BLOCK,BLOCK);
  dim3 grid(N/BLOCK,N/BLOCK);

  matrixMul<<<grid,block>>>(dMatA,dMatB,dMatC);
  hipDeviceSynchronize();

  hMatC=(int*)malloc(matrixSize);
  hipMemcpy(hMatC,dMatC,matrixSize,hipMemcpyDeviceToHost);

  free(hMatA);
  free(hMatB);
  free(hMatC);
  hipFree(dMatA);
  hipFree(dMatB);
  hipFree(dMatC);

  hipDeviceReset();
  cutilExit(argc,argv);
}

__global__ void
matrixMul(int* inMatA,int* inMatB,int* inMatC){
  int col=blockIdx.x * blockDim.x + threadIdx.x;
  int row=blockIdx.y * blockDim.y + threadIdx.y;
  int scan;
  int target=0;

  for(scan=0;scan<N;scan++){
    target+=inMatA[col*N+scan]*inMatB[scan*N+row];
    __syncthreads();
  }

  inMatC[col*N+row]=target;
}
